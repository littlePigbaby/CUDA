#include "hip/hip_runtime.h"
// rmsnorm_kernel.cu

#include <torch/extension.h>
#include <hip/hip_runtime.h>

__global__ void rmsnorm_kernel(float* input, float* output, int size, float epsilon) {
    extern __shared__ float shared_mem[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int lane = threadIdx.x;

    shared_mem[lane] = (tid < size) ? input[tid] * input[tid] : 0.0f;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        if (lane < offset) {
            shared_mem[lane] += shared_mem[lane + offset];
        }
        __syncthreads();
    }

    float rms = sqrtf(shared_mem[0] / size + epsilon);

    if (tid < size) {
        output[tid] = input[tid] / rms;
    }
}

void rmsnorm_cuda(torch::Tensor input, torch::Tensor output, float epsilon) {
    int size = input.size(0);
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    rmsnorm_kernel<<<numBlocks, blockSize, blockSize * sizeof(float)>>>(
        input.data_ptr<float>(), output.data_ptr<float>(), size, epsilon);

    hipDeviceSynchronize();
}
